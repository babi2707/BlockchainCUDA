#include "hip/hip_runtime.h"
#include "Block.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>  // Para std::hex e std::setw
#include <sstream>  // Para std::stringstream

// Variável global do dispositivo para indicar se a solução foi encontrada
__device__ bool dev_found = false;

// Função dummy para simular o cálculo do SHA-256
__device__ void dummy_sha256(const char *input, char *output)
{
    int inputLength = 0;
    while (input[inputLength] != '\0')
    {
        ++inputLength;
    }

    // Simular o cálculo do hash
    for (int i = 0; i < 64; i++)
    {
        output[i] = (input[i % inputLength] + i) % 256;
    }
    output[64] = '\0';
}

// Função _CalculateHash vazia (não está sendo usada)
__device__ void _CalculateHash(char *output, uint32_t index, const char *prevHash, time_t tTime, const char *data, uint32_t nonce)
{
}

// Função para comparar duas strings no dispositivo
__device__ int dev_strncmp(const char *str1, const char *str2, size_t num)
{
    for (size_t i = 0; i < num; i++)
    {
        if (str1[i] != str2[i])
        {
            return str1[i] - str2[i];
        }
        if (str1[i] == '\0')
        {
            return 0;
        }
    }
    return 0;
}

// Função para copiar uma string no dispositivo
__device__ void dev_strncpy(char *dest, const char *src, size_t num)
{
    for (size_t i = 0; i < num; i++)
    {
        dest[i] = src[i];
        if (src[i] == '\0')
        {
            break;
        }
    }
    if (num > 0)
    {
        dest[num - 1] = '\0';
    }
}

// Kernel de mineração de bloco
__global__ void MineBlockKernel(uint32_t index, const char* prevHash, time_t tTime, const char* data, uint32_t difficulty, char* result) {
    // Calcula o nonce inicial baseado no índice do bloco e da thread
    uint32_t nonce = blockIdx.x * blockDim.x + threadIdx.x;
    char target[65];

    // Prepara a string target de zeros baseada na dificuldade
    for (uint32_t i = 0; i < difficulty; ++i) {
        target[i] = '0';
    }
    target[difficulty] = '\0';
    char hash[65];

    while (!dev_found) {
        // Construir o input para o hash diretamente
        int pos = 0;

        // Copiar prevHash para hash
        for (int i = 0; i < 64 && prevHash[i] != '\0'; ++i) {
            hash[pos++] = prevHash[i];
        }

        // Copiar data para hash
        for (int i = 0; i < 64 && data[i] != '\0'; ++i) {
            hash[pos++] = data[i];
        }

        // Copiar tTime para hash (usando reinterpret_cast para tratar como char*)
        char* tTimeChar = reinterpret_cast<char*>(&tTime);
        for (int i = 0; i < sizeof(time_t); ++i) {
            hash[pos++] = tTimeChar[i];
        }

        // Copiar index para hash (usando reinterpret_cast para tratar como char*)
        char* indexChar = reinterpret_cast<char*>(&index);
        for (int i = 0; i < sizeof(uint32_t); ++i) {
            hash[pos++] = indexChar[i];
        }

        // Copiar nonce para hash (usando reinterpret_cast para tratar como char*)
        char* nonceChar = reinterpret_cast<char*>(&nonce);
        for (int i = 0; i < sizeof(uint32_t); ++i) {
            hash[pos++] = nonceChar[i];
        }

        // Calcular o hash usando dummy_sha256
        dummy_sha256(hash, hash);

        // Verificar se o hash atende à dificuldade
        if (dev_strncmp(hash, target, difficulty) == 0) {
            // Se o hash atende à dificuldade, tenta marcar como encontrado usando atomicExch
            if (atomicExch(reinterpret_cast<unsigned int*>(&dev_found), 1u) == 0) {
                dev_strncpy(result, hash, 64); // Copiar apenas os primeiros 64 caracteres
                result[64] = '\0'; // Garantir que o resultado seja uma string terminada por '\0'
            }
        }
        // Incrementa o nonce para o próximo cálculo
        nonce += gridDim.x * blockDim.x;
    }
}

// Função para chamar o kernel de mineração de bloco na GPU
extern "C" void MineBlockGPU(uint32_t index, const char* prevHash, time_t tTime, const char* data, uint32_t difficulty, char* result) {
    char* dev_result;
    hipMalloc((void**)&dev_result, 65); // Alocar espaço para 65 caracteres

    // Lançar o kernel com 7 blocos e 12 threads por bloco
    MineBlockKernel<<<7, 12>>>(index, prevHash, tTime, data, difficulty, dev_result);
    hipDeviceSynchronize(); // Sincronizar a execução do dispositivo

    // Copiar o resultado de volta para a memória do host
    hipMemcpy(result, dev_result, 65, hipMemcpyDeviceToHost);

    // Garantir que o resultado seja uma string terminada por '\0'
    result[64] = '\0';

    // Liberar memória do dispositivo
    hipFree(dev_result);
}

// Implementação da função MineBlock da classe Block
void Block::MineBlock(unsigned int difficulty) {
    char result[65];
    MineBlockGPU(index, previousHash.c_str(), timestamp, data.c_str(), difficulty, result);
    std::cout << "Mining block " << index << "..." << std::endl;

    // Converter o hash binário para uma string hexadecimal
    std::stringstream ss;
    for (int i = 0; i < 32; ++i) {
        ss << std::setw(2) << std::setfill('0') << std::hex << (int)(unsigned char)result[i];
    }
    std::string hexHash = ss.str();

    std::cout << "Block mined: " << hexHash << std::endl;
}

// Definição do construtor de Block
Block::Block(unsigned int index, const std::string &data)
    : index(index), data(data)
{
    this->timestamp = std::time(nullptr);
    this->previousHash = "previous_hash_value";
    this->hash = "";
}
